#include "hip/hip_runtime.h"
#include "ELLPACKCudaSolver.h"
#include <host_defines.h>
#include <>
#include <hip/hip_runtime.h>

__global__ void ellpackKernel(INDEXING_TYPE rows, INDEXING_TYPE max_row_non_zeros, INDEXING_TYPE *JA, FLOATING_TYPE *AS, FLOATING_TYPE *b, FLOATING_TYPE *x) {
  //TODO: Add ellpackKernel	
  int row = blockDim.x * blockIdx.x + threadIdx.x;

  if(row < rows) {
    FLOATING_TYPE dot = 0;
    for(int i = 0; i < max_row_non_zeros; i++) {
      auto index = row * max_row_non_zeros + i;
      int col = JA[index];
      FLOATING_TYPE val = AS[index];
      if(val != 0)
        dot += val * b[col];
    }
    x[row] += dot;
  }

}

sspp::representations::Output sspp::tools::solvers::ELLPACKCudaSolver::Solve(sspp::representations::ELLPACK & ellpack, std::vector<FLOATING_TYPE> & b) {
  std::vector<FLOATING_TYPE> x(ellpack.GetRows());
  //TODO: handle cudaStatus exceptions
  hipError_t cudaStatus;
  FLOATING_TYPE *device_x, *device_b, *device_as;
  INDEXING_TYPE *device_ja;

  cudaStatus = hipMalloc(&device_x, sizeof(FLOATING_TYPE) * x.size());
  cudaStatus = hipMalloc(&device_b, sizeof(FLOATING_TYPE) * b.size());
  cudaStatus = hipMalloc(&device_as, sizeof(FLOATING_TYPE) * ellpack.GetAS().size());
  cudaStatus = hipMalloc(&device_ja, sizeof(INDEXING_TYPE) * ellpack.GetJA().size());

  cudaStatus = hipMemcpy(device_x, &x[0], sizeof(FLOATING_TYPE) * x.size(), hipMemcpyHostToDevice);
  cudaStatus = hipMemcpy(device_b, &b[0], sizeof(FLOATING_TYPE) * b.size(), hipMemcpyHostToDevice);
  cudaStatus = hipMemcpy(device_as, &ellpack.GetAS()[0], sizeof(FLOATING_TYPE) * ellpack.GetAS().size(), hipMemcpyHostToDevice);
  cudaStatus = hipMemcpy(device_ja, &ellpack.GetJA()[0], sizeof(INDEXING_TYPE) * ellpack.GetJA().size(), hipMemcpyHostToDevice);

  ellpackKernel << <ellpack.GetRows(), 1 >> > (ellpack.GetRows(), ellpack.GetMaxRowNonZeros(), device_ja, device_as, device_b, device_x);

  cudaStatus = hipMemcpy(&x[0], device_x, sizeof(FLOATING_TYPE) * x.size(), hipMemcpyDeviceToHost);

  hipFree(device_x);
  hipFree(device_b);
  hipFree(device_as);
  hipFree(device_ja);

  return sspp::representations::Output(x);
}
