#include "hip/hip_runtime.h"
#include "ELLPACKCudaSolver.h"
#include <host_defines.h>
#include <>
#include <hip/hip_runtime.h>

__global__ void ellpackKernel(sspp::representations::ELLPACK & ellpack, FLOATING_TYPE* b, FLOATING_TYPE *x) {
  //TODO: Add ellpackKernel	
  int row = blockDim.x * blockIdx.x + threadIdx.x;

  if(row < ellpack.M) {
    FLOATING_TYPE dot = 0;
    for(int i = 0; i < ellpack.MAXNZ; i++) {
      int col = ellpack.JA[row][i];
      FLOATING_TYPE val = ellpack.AS[row][i];
      if(val != 0)
        dot += val * b[col];
    }
    x[row] += dot;
  }

}

sspp::representations::Output sspp::tools::solvers::ELLPACKCudaSolver::solve(sspp::representations::ELLPACK & ellpack, FLOATING_TYPE * b) {
  FLOATING_TYPE *x = new FLOATING_TYPE[ellpack.M];
  hipError_t cudaStatus;
  sspp::representations::ELLPACK *hEllpackPtr = &ellpack;
  sspp::representations::ELLPACK *dEllpackPtr;
  FLOATING_TYPE** h_AS = hEllpackPtr->AS;
  int** h_JA = hEllpackPtr->JA;

  int *d_JA;
  FLOATING_TYPE *d_AS, *d_B, *d_X;

  hipMalloc(&d_JA, sizeof(int) * ellpack.M * ellpack.MAXNZ);
  hipMemcpy(d_JA, h_JA, sizeof(int) * ellpack.M * ellpack.MAXNZ, hipMemcpyHostToDevice);

  hEllpackPtr->JA = &d_JA;

  hipMalloc(&d_AS, sizeof(FLOATING_TYPE) * ellpack.M * ellpack.MAXNZ);
  hipMemcpy(d_AS, h_AS, sizeof(FLOATING_TYPE) * ellpack.M * ellpack.MAXNZ, hipMemcpyHostToDevice);

  hEllpackPtr->AS = &d_AS;

  hipMalloc(&d_B, sizeof(FLOATING_TYPE) * ellpack.M);
  hipMemcpy(d_B, b, sizeof(FLOATING_TYPE) * ellpack.M * ellpack.MAXNZ, hipMemcpyHostToDevice);

  hipMalloc(&d_X, sizeof(FLOATING_TYPE) * ellpack.M);

  ellpackKernel << <ellpack.M, 1 >> > (ellpack, d_B, d_X);

  hipMemcpy(x, d_X, sizeof(FLOATING_TYPE)*ellpack.M, hipMemcpyDeviceToHost);

  return sspp::representations::Output(ellpack.M, x);
}
