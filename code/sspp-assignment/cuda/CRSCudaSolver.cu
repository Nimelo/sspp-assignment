#include "hip/hip_runtime.h"
#include "CRSCudaSolver.h"
#include <host_defines.h>
#include <>
#include <hip/hip_runtime.h>

__global__ void crsKernel(unsigned rows, unsigned *irp, unsigned *ja, float *as, float* b, float *x) {
  int row = blockDim.x * blockIdx.x + threadIdx.x;
  if(row < rows) {
    float dot = 0;
    int rowStart = irp[row],
      rowEnd = irp[row + 1];

    for(int i = rowStart; i < rowEnd; i++) {
      dot += as[i] * b[ja[i]];
    }

    x[row] += dot;
  }
}

sspp::common::Output<float> sspp::tools::solvers::CRSCudaSolver::Solve(common::CRS<float>& crs, std::vector<float>& b) {
  std::vector<float> x(crs.GetRows());
  hipError_t cudaStatus;
  unsigned *device_irp, *device_ja;
  float *device_as, *device_b, *device_x;
  //TODO: handle cudaStatus exceptions
  cudaStatus = hipMalloc(&device_irp, sizeof(unsigned) * crs.GetRowStartIndexes().size());
  cudaStatus = hipMalloc(&device_ja, sizeof(unsigned) * crs.GetColumnIndices().size());
  cudaStatus = hipMalloc(&device_as, sizeof(float) * crs.GetValues().size());
  cudaStatus = hipMalloc(&device_b, sizeof(float) * b.size());
  cudaStatus = hipMalloc(&device_x, sizeof(float) * x.size());

  cudaStatus = hipMemcpy(device_irp, &crs.GetRowStartIndexes()[0], sizeof(unsigned) * crs.GetRowStartIndexes().size(), hipMemcpyHostToDevice);
  cudaStatus = hipMemcpy(device_ja, &crs.GetColumnIndices()[0], sizeof(unsigned) * crs.GetColumnIndices().size(), hipMemcpyHostToDevice);
  cudaStatus = hipMemcpy(device_as, &crs.GetValues()[0], sizeof(float) * crs.GetValues().size(), hipMemcpyHostToDevice);
  cudaStatus = hipMemcpy(device_b, &b[0], sizeof(float) * b.size(), hipMemcpyHostToDevice);
  cudaStatus = hipMemcpy(device_x, &x[0], sizeof(float) * x.size(), hipMemcpyHostToDevice);
  unsigned int x_dimension = crs.GetRows() > 1024 ? 1024 : crs.GetRows();
  crsKernel << <x_dimension, 1 >> > (crs.GetRows(), device_irp, device_ja, device_as, device_b, device_x);

  cudaStatus = hipMemcpy(&x[0], device_x, sizeof(float) * x.size(), hipMemcpyDeviceToHost);

  hipFree(device_irp);
  hipFree(device_ja);
  hipFree(device_as);
  hipFree(device_b);
  hipFree(device_x);

  return common::Output<float>(x);
}
