#include "hip/hip_runtime.h"
#include "CSRCudaSolver.h"
#include <host_defines.h>
#include <>
#include <hip/hip_runtime.h>

__global__ void csrKernel(INDEXING_TYPE rows, INDEXING_TYPE *irp, INDEXING_TYPE *ja, FLOATING_TYPE *as, FLOATING_TYPE* b, FLOATING_TYPE *x) {
  int row = blockDim.x * blockIdx.x + threadIdx.x;
  if(row < rows) {
    FLOATING_TYPE dot = 0;
    int rowStart = irp[row],
      rowEnd = irp[row + 1];

    for(int i = rowStart; i < rowEnd; i++) {
      dot += as[i] * b[ja[i]];
    }

    x[row] += dot;
  }
}

sspp::representations::Output sspp::tools::solvers::CSRCudaSolver::Solve(sspp::representations::CSR & csr, std::vector<FLOATING_TYPE> & b) {
  std::vector<FLOATING_TYPE> x(csr.GetRows());
  hipError_t cudaStatus;
  INDEXING_TYPE *device_irp, *device_ja;
  FLOATING_TYPE *device_as, *device_b, *device_x;
  //TODO: handle cudaStatus exceptions
  cudaStatus = hipMalloc(&device_irp, sizeof(INDEXING_TYPE) * csr.GetIRP().size());
  cudaStatus = hipMalloc(&device_ja, sizeof(INDEXING_TYPE) * csr.GetJA().size());
  cudaStatus = hipMalloc(&device_as, sizeof(FLOATING_TYPE) * csr.GetAS().size());
  cudaStatus = hipMalloc(&device_b, sizeof(FLOATING_TYPE) * b.size());
  cudaStatus = hipMalloc(&device_x, sizeof(FLOATING_TYPE) * x.size());

  cudaStatus = hipMemcpy(device_irp, &csr.GetIRP()[0], sizeof(INDEXING_TYPE) * csr.GetIRP().size(), hipMemcpyHostToDevice);
  cudaStatus = hipMemcpy(device_ja, &csr.GetJA()[0], sizeof(INDEXING_TYPE) * csr.GetJA().size(), hipMemcpyHostToDevice);
  cudaStatus = hipMemcpy(device_as, &csr.GetAS()[0], sizeof(FLOATING_TYPE) * csr.GetAS().size(), hipMemcpyHostToDevice);
  cudaStatus = hipMemcpy(device_b, &b[0], sizeof(FLOATING_TYPE) * b.size(), hipMemcpyHostToDevice);
  cudaStatus = hipMemcpy(device_x, &x[0], sizeof(FLOATING_TYPE) * x.size(), hipMemcpyHostToDevice);

  csrKernel << <csr.GetRows(), 1 >> > (csr.GetRows(), device_irp, device_ja, device_as, device_b, device_x);

  cudaStatus = hipMemcpy(&x[0], device_x, sizeof(FLOATING_TYPE) * x.size(), hipMemcpyDeviceToHost);

  hipFree(device_irp);
  hipFree(device_ja);
  hipFree(device_as);
  hipFree(device_b);
  hipFree(device_x);

  return representations::Output(x);
}
